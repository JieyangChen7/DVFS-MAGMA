#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.7.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2015

       @generated from zcaxpycp.cu mixed zc -> ds, Fri Sep 11 18:29:19 2015

*/
#include "common_magma.h"

#define NB 64

// adds   x += r (including conversion to double)  --and--
// copies w = b
// each thread does one index, x[i] and w[i]
__global__ void
dsaxpycp_kernel(
    int m, float *r, double *x,
    const double *b, double *w )
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_D_ADD( x[i], (double)( r[i] ) );
        w[i] = b[i];
    }
}


// ----------------------------------------------------------------------
// adds   x += r (including conversion to double)  --and--
// copies w = b
extern "C" void
magmablas_dsaxpycp_q(
    magma_int_t m,
    magmaFloat_ptr r,
    magmaDouble_ptr x,
    magmaDouble_const_ptr b,
    magmaDouble_ptr w,
    magma_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    dsaxpycp_kernel <<< grid, threads, 0, queue >>> ( m, r, x, b, w );
}


extern "C" void
magmablas_dsaxpycp(
    magma_int_t m,
    magmaFloat_ptr r,
    magmaDouble_ptr x,
    magmaDouble_const_ptr b,
    magmaDouble_ptr w)
{
    magmablas_dsaxpycp_q( m, r, x, b, w, magma_stream );
}
