#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.7.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2015
       
       @author Mark Gates
       @author Tingxing Dong
       @author Azzam Haidar

       @generated from zgemv_fermi.cu normal z -> d, Fri Sep 11 18:29:22 2015
*/
#include "common_magma.h"
#include "commonblas_d.h"
#include "magma_templates.h"

#define PRECISION_d

#include "gemv_template_device.cuh"

#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"

#define version(s,v) s ## _V_ ## v


//////////////////////////////////////////////////////////////////////////////////////////
// NoTrans kernel
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
__global__ void
dgemvn_template_kernel_fermi(
    int m, int n, double alpha,
    const double * __restrict__ A, int lda,
    const double * __restrict__ x, int incx, double beta,
    double       *y, int incy)
{
#if (__CUDA_ARCH__ >= 200)

    gemvn_template_device<double, DIM_X, DIM_Y, TILE_SIZE>(m, n, alpha, A, lda, x, incx, beta, y, incy);
#endif /* (__CUDA_ARCH__ >= 200) */
}
//////////////////////////////////////////////////////////////////////////////////////////
// Trans/ConjTans kernel
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE, magma_trans_t trans>
__global__ void
dgemvc_template_kernel_fermi(
    int m, int n, double alpha,
    const double * __restrict__ A, int lda,
    const double * __restrict__ x, int incx, double beta,
    double       *y, int incy)
{
#if (__CUDA_ARCH__ >= 200)
    gemvc_template_device< double, DIM_X, DIM_Y, TILE_SIZE, trans >(m, n, alpha, A, lda, x, incx, beta, y, incy);
#endif /* (__CUDA_ARCH__ >= 200) */
}
//////////////////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////////////////////
// NoTrans CPU driver
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
void
dgemvn_template_fermi(
    magma_int_t m, magma_int_t n, double alpha,
    const double * __restrict__ A, magma_int_t lda,
    const double * __restrict__ x, magma_int_t incx, double beta,
    double       *y, magma_int_t incy, magma_queue_t queue)
{
    dim3 grid( magma_ceildiv(m, TILE_SIZE) );
    dim3 threads( DIM_X, DIM_Y, 1 );

    dgemvn_template_kernel_fermi<DIM_X, DIM_Y, TILE_SIZE><<< grid, threads, 0, queue >>>(m, n, alpha, A, lda, x, incx, beta, y, incy);
}
//////////////////////////////////////////////////////////////////////////////////////////
// Trans/ConjTans CPU driver
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
void
dgemvc_template_fermi(
    magma_trans_t trans, magma_int_t m, magma_int_t n, double alpha,
    const double * __restrict__ A, magma_int_t lda,
    const double * __restrict__ x, magma_int_t incx, double beta,
    double       *y, magma_int_t incy, magma_queue_t queue)
{
    dim3 grid    ( 1,  magma_ceildiv(n, TILE_SIZE),  1 );
    dim3 threads ( DIM_X, DIM_Y, 1 );

    if (trans == MagmaConjTrans)
    {
        dgemvc_template_kernel_fermi< DIM_X, DIM_Y, TILE_SIZE, MagmaConjTrans ><<< grid, threads, 0, queue >>>(m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
    else
    {
        dgemvc_template_kernel_fermi< DIM_X, DIM_Y, TILE_SIZE, MagmaTrans ><<< grid, threads, 0, queue >>>(m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
}



//////////////////////////////////////////////////////////////////////////////////////////

/**
    Purpose
    -------
    DGEMV performs one of the matrix-vector operations
    
        y := alpha*A*x    + beta*y,   or
        y := alpha*A**T*x + beta*y,   or
        y := alpha*A**H*x + beta*y,
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    trans   magma_trans_t
            On entry, TRANS specifies the operation to be performed as
            follows:
      -     = MagmaNoTrans:    y := alpha*A  *x + beta*y
      -     = MagmaTrans:      y := alpha*A^T*x + beta*y
      -     = MagmaConjTrans:  y := alpha*A^H*x + beta*y

    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A
 
    @param[in]
    alpha   DOUBLE_PRECISION
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      DOUBLE_PRECISION array of dimension ( LDDA, n ) on the GPU.
   
    @param[in]
    ldda    INTEGER
            LDDA specifies the leading dimension of A.

    @param[in]
    dx      DOUBLE_PRECISION array of dimension
            n if trans == MagmaNoTrans
            m if trans == MagmaTrans or MagmaConjTrans
     
    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.
  
    @param[in]
    beta    DOUBLE_PRECISION
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      DOUBLE_PRECISION array of dimension
            m if trans == MagmaNoTrans
            n if trans == MagmaTrans or MagmaConjTrans

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_dblas2
    ********************************************************************/
extern "C" void
magmablas_dgemv_q(
    magma_trans_t trans, magma_int_t m, magma_int_t n, 
    double alpha,
    magmaDouble_const_ptr dA, magma_int_t ldda,
    magmaDouble_const_ptr dx, magma_int_t incx,
    double beta,
    magmaDouble_ptr dy, magma_int_t incy, 
    magma_queue_t queue)
{
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }

    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200  ) {
        // --------------------
        // call CUDA ARCH 1.x version
        // magmablas for [sd] precisions, cublas for [zc] precisions.
        #if defined(PRECISION_z) || defined(PRECISION_c)
        magma_dgemv( trans, m, n, alpha, dA, ldda, dx, incx, beta, dy, incy );
        #else
        magmablas_dgemv_tesla( trans, m, n, alpha, dA, ldda, dx, incx, beta, dy, incy );
        #endif
        return;
    }
    
    
    // --------------------
    // CUDA ARCH 2.x (Fermi) version
    if ( trans == MagmaNoTrans ) {
        if (m <= 256) {
            dgemvn_template_fermi<version(N, 137)>
                ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
        }
        else {
            dgemvn_template_fermi<version(N, 140)>
                ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
        }
    }
    else {
        dgemvc_template_fermi<version(T, 189)>
            ( trans, m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
    }
}


extern "C" void
magmablas_dgemv(
    magma_trans_t trans, magma_int_t m, magma_int_t n, double alpha,
    magmaDouble_const_ptr dA, magma_int_t ldda,
    magmaDouble_const_ptr dx, magma_int_t incx,
    double beta,
    magmaDouble_ptr dy, magma_int_t incy)
{
    magmablas_dgemv_q( trans, m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, magma_stream);
}
