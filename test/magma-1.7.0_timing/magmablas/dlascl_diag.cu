#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.7.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2015

       @generated from zlascl_diag.cu normal z -> d, Fri Sep 11 18:29:20 2015
*/
#include "common_magma.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
dlascl_diag_lower(int m, int n, double const* D, int ldd, 
                                double*       A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for (int j=0; j < n; j++ )
            A[j*lda] /= D[j + j*ldd];
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
dlascl_diag_upper(int m, int n, double const* D, int ldd, 
                                double*       A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for (int j=0; j < n; j++ )
            A[j*lda] /= D[ind + ind*ldd];
    }
}


/**
    Purpose
    -------
    DLASCL_DIAG scales the M by N real matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    @param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaFull:   full matrix.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in]
    dD      DOUBLE PRECISION vector, dimension (LDDD,M)
            The matrix storing the scaling factor on its diagonal. 

    @param[in]
    lddd    INTEGER
            The leading dimension of the array D.  

    @param[in,out]
    dA      DOUBLE PRECISION array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlascl_diag_q(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr dD, magma_int_t lddd, 
    magmaDouble_ptr       dA, magma_int_t ldda, 
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper && type != MagmaFull )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    //else if ( ldda < max(1,m) )
    //    *info = -5;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( magma_ceildiv( m, NB ) );
    dim3 threads( NB );
    
    if (type == MagmaLower) {
        dlascl_diag_lower <<< grid, threads, 0, queue >>> (m, n, dD, lddd, dA, ldda);
    }
    else if (type == MagmaUpper) {
        dlascl_diag_upper <<< grid, threads, 0, queue >>> (m, n, dD, lddd, dA, ldda);
    }
}


/**
    @see magmablas_dlascl2_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlascl_diag(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr dD, magma_int_t lddd, 
    magmaDouble_ptr       dA, magma_int_t ldda, 
    magma_int_t *info )
{
    magmablas_dlascl_diag_q( type, m, n, dD, lddd, dA, ldda, magma_stream, info );
}
