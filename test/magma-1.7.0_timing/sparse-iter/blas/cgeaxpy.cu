#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.7.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2015

       @generated from zgeaxpy.cu normal z -> c, Fri Sep 11 18:29:42 2015

*/
#include "common_magma.h"
#include "common_magmasparse.h"

#define BLOCK_SIZE 256


// axpy kernel for matrices stored in the MAGMA format
__global__ void 
cgeaxpy_kernel( 
    int num_rows, 
    int num_cols, 
    magmaFloatComplex alpha, 
    magmaFloatComplex * dx, 
    magmaFloatComplex beta, 
    magmaFloatComplex * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if( row<num_rows ){
        for( j=0; j<num_cols; j++ ){
            int idx = row + j*num_rows;
            dy[ idx ] = alpha * dx[ idx ] + beta * dy[ idx ];
        }
    }
}

/**
    Purpose
    -------
    
    This routine computes Y = alpha *  X + beta * Y on the GPU.
    The input format is a dense matrix (vector block) stored in 
    magma_c_matrix format.
    
    Arguments
    ---------

    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier.
                
    @param[in]
    X           magma_c_matrix
                input/output matrix Y.
                
    @param[in]
    beta        magmaFloatComplex
                scalar multiplier.
                
    @param[in,out]
    Y           magma_c_matrix*
                input matrix X.
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" 
magma_int_t
magma_cgeaxpy(
    magmaFloatComplex alpha,
    magma_c_matrix X,
    magmaFloatComplex beta,
    magma_c_matrix *Y,
    magma_queue_t queue )
{
    int m = X.num_rows;
    int n = X.num_cols;
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    cgeaxpy_kernel<<< grid, threads, 0, queue >>>
                    ( m, n, alpha, X.dval, beta, Y->dval );
                    
    return MAGMA_SUCCESS;
}
