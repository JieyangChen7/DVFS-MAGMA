#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.7.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2015

       @generated from zgeaxpy.cu normal z -> d, Fri Sep 11 18:29:42 2015

*/
#include "common_magma.h"
#include "common_magmasparse.h"

#define BLOCK_SIZE 256


// axpy kernel for matrices stored in the MAGMA format
__global__ void 
dgeaxpy_kernel( 
    int num_rows, 
    int num_cols, 
    double alpha, 
    double * dx, 
    double beta, 
    double * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if( row<num_rows ){
        for( j=0; j<num_cols; j++ ){
            int idx = row + j*num_rows;
            dy[ idx ] = alpha * dx[ idx ] + beta * dy[ idx ];
        }
    }
}

/**
    Purpose
    -------
    
    This routine computes Y = alpha *  X + beta * Y on the GPU.
    The input format is a dense matrix (vector block) stored in 
    magma_d_matrix format.
    
    Arguments
    ---------

    @param[in]
    alpha       double
                scalar multiplier.
                
    @param[in]
    X           magma_d_matrix
                input/output matrix Y.
                
    @param[in]
    beta        double
                scalar multiplier.
                
    @param[in,out]
    Y           magma_d_matrix*
                input matrix X.
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" 
magma_int_t
magma_dgeaxpy(
    double alpha,
    magma_d_matrix X,
    double beta,
    magma_d_matrix *Y,
    magma_queue_t queue )
{
    int m = X.num_rows;
    int n = X.num_cols;
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    dgeaxpy_kernel<<< grid, threads, 0, queue >>>
                    ( m, n, alpha, X.dval, beta, Y->dval );
                    
    return MAGMA_SUCCESS;
}
